
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(void)
{
    int numberOfBlock = 32;
    int threadsPerBlock = 8;
    int vectorSize = numberOfBlock * threadsPerBlock;
    int size = vectorSize * sizeof(int);

    int *a = (int *)malloc(size);
    int *b = (int *)malloc(size);
    int *c = (int *)malloc(size);

    for (int i = 0; i < vectorSize; ++i)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    int *da, *db, *dc;

    hipMallocManaged(&da, size);
    hipMallocManaged(&db, size);
    hipMallocManaged(&dc, size);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dc, c, size, hipMemcpyHostToDevice);

    add<<<numberOfBlock, threadsPerBlock>>>(da, db, dc);

    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < vectorSize; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");

    return 0;
}