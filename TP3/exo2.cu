
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multiply(int *a, int *b, int *c, int vectorSize, int nbRow)
{

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int row = index / nbRow;
    int col = index % nbRow;

    int rslt = 0;
    for (int i = 0; i < nbRow; i++)
    {
        rslt += a[row * nbRow + i] * b[i * nbRow + col];
    }

    c[index] = rslt;
}

int main(void)
{
    int numberOfBlock = 32;
    int threadsPerBlock = 8;
    int rsltVectorSize = numberOfBlock * threadsPerBlock;
    int vectorSize = sqrt(rsltVectorSize) * 8;
    int nbRow = sqrt(rsltVectorSize);
    int size = vectorSize * sizeof(int);
    int rslSize = rsltVectorSize * sizeof(int);

    int *a = (int *)malloc(size);
    int *b = (int *)malloc(size);
    int *c = (int *)malloc(rslSize);

    for (int i = 0; i < nbRow * 8; ++i)
    {
        a[i] = i;
        b[i] = i;
    }

    for (int i = 0; i < rsltVectorSize; ++i)
    {
        c[i] = 0;
    }

    int *da, *db, *dc;

    hipMallocManaged(&da, size);
    hipMallocManaged(&db, size);
    hipMallocManaged(&dc, rslSize);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dc, c, rslSize, hipMemcpyHostToDevice);

    multiply<<<numberOfBlock, threadsPerBlock>>>(da, db, dc, vectorSize, nbRow); // 128 = vectorSize / 2

    hipMemcpy(c, dc, rslSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < rsltVectorSize; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\n");

    return 0;
}